 
#include <hip/hip_runtime.h>
#include <stdio.h>
 #define N 10000000
 #define numThread 2 
 #define numBlock 200  

 __global__ void add( int *a, int *b, int *c ) {
 
     int tid = blockDim.x * blockIdx.x + threadIdx.x;
     while (tid < N) {
         c[tid] = a[tid] + b[tid];      
         tid += blockDim.x;       
                                  
     }
 }
 
 
 int main( void ) {
     int *a, *b, *c;               // The arrays on the host CPU machine
     int *dev_a, *dev_b, *dev_c;   // The arrays for the GPU device
 
     a = (int*)malloc( N * sizeof(int) );
     b = (int*)malloc( N * sizeof(int) );
     c = (int*)malloc( N * sizeof(int) );
 
     for (int i=0; i<N; i++) {
         a[i] = i;
         b[i] = i;
     }
 
      hipMalloc( (void**)&dev_a, N * sizeof(int) );
      hipMalloc( (void**)&dev_b, N * sizeof(int) );
      hipMalloc( (void**)&dev_c, N * sizeof(int) );
      hipMemcpy( dev_a, a, N * sizeof(int),
                               hipMemcpyHostToDevice );
      hipMemcpy( dev_b, b, N * sizeof(int),
                               hipMemcpyHostToDevice );
     add<<<numBlock,numThread>>>( dev_a, dev_b, dev_c );
     hipMemcpy( c, dev_c, N * sizeof(int),
                               hipMemcpyDeviceToHost );
     bool success = true;
     int total=0;
     printf("Checking %d values in the array.\n", N);
     for (int i=0; i<N; i++) {
         if ((a[i] + b[i]) != c[i]) {
             printf( "Error:  %d + %d != %d\n", a[i], b[i], c[i] );
             success = false;
         }
         total += 1;
     }
     if (success)  printf( "We did it, %d values correct!\n", total );
     free( a );
     free( b );
     free( c );
      hipFree( dev_a );
      hipFree( dev_b );
      hipFree( dev_c );
     return 0;
 }
